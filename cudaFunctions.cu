#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <stdio.h>
#include <iostream>
#include <string.h>
#include <limits.h>
#include "cudaHeader.h"

__device__ __constant__ char hashA[ALPHABET][ITEM_MAX_LEN] = {"ST","","","NEQ","NDQK","YWMIL","","QRKYN","MLFV","","NEQHR","MIVF","ILVF","DEQHK","","","NDEHRK","QHK","TA","SA","","MIL","FY","","FWH",""};
__device__ __constant__ char hashB[ALPHABET][ITEM_MAX_LEN] = {"SGTVCP","","SA","SGNEQHK","NQHRKDS","HYVLIM","SAND","NEQRKDFY","FVLM","","NEQHRDST","FVIM","FVLI","SGDEQHRKT","","STA","NEHRKDS","NEQHK","AGNDTPCKEQ","AVSPNK","","TAFLIM","","","HF",""};


__device__ int check_group_similarity_cuda(char hashX[ALPHABET][ITEM_MAX_LEN], char seq1_char, char seq2_char)
{	

	int i;
	int letter_index = seq1_char - 'A';
	if(hashX[letter_index] == "")
		return 0;

	for(i =0 ; i<ITEM_MAX_LEN ;i++)
	{
		if(hashX[letter_index][i] == seq2_char)
			return 1;
	}
	return 0;
}


__global__ void thread_score_offset_calc(char* seq1, char* seq2 , int* lenght_seq2 , int* weights, int* start, int* end, int* res)
{
    int total_mutations = *lenght_seq2;
    int score = 0;
    int index1 = 0;

    int thread_id = threadIdx.x + blockIdx.x * blockDim.x;
    int mutant =  thread_id;
    int thread_index_for_res_array = thread_id * RES_CACHE_LEN; 

    if(mutant <= total_mutations && mutant !=0)
    {
        for (int offset = *start; offset < *end; offset++) 
        {
            index1 = offset;
            for (int index2 = 0; index2 < *lenght_seq2; index2++)
            {
                if (index2 == mutant)
                    index1++;
                
                if (seq1[index1] == seq2[index2])
                    score += weights[DOLLAR];
                
                else if (check_group_similarity_cuda(hashA, seq1[index1], seq2[index2]))
                    score -= weights[PRECENT];
                
                else if (check_group_similarity_cuda(hashB, seq1[index1], seq2[index2]))
                    score -= weights[HASH];
                
                else
                    score -= weights[SPACE];
                index1++;
            }
            
            if(score > res[thread_index_for_res_array])
            {
                res[thread_index_for_res_array] = score;
                res[thread_index_for_res_array + OFFSET] = offset;
                res[thread_index_for_res_array + MUTANT] = mutant;
            }
            score = 0;
        }
    } 
}

//CHECK FUNCTION
void checkStatus(hipError_t cudaStatus, std::string err)
{
    if(cudaStatus != hipSuccess)
    {
        std::cout << err << std::endl;
        exit(1);

    }
}


int* cuda_score_offset_mutant(char* seq1, char* seq2 , int* weights, int offset_start, int offset_end)
{
    hipError_t cudaStatus;

    // EACH THREAD CHECK ALL OFFSETS FOR A SINGLE MUTATION
    int* optimal_res = (int*)malloc(sizeof(int)*RES_CACHE_LEN);

    int seq2_lenght = (strlen(seq2));
    int total_mutations = seq2_lenght;
    int num_of_blocks = (total_mutations / THREADS_PER_BLOCK);
    if (total_mutations % THREADS_PER_BLOCK != 0)
        num_of_blocks ++;
    
    // POINTERS FOR CUDA MEM
    char* cuda_seq1, *cuda_seq2;
    int* cuda_weights, *cuda_offset_start, *cuda_offset_end ,*cuda_seq2_len;
    int* cuda_res_array, *res_array;

    int res_array_size = total_mutations * (RES_CACHE_LEN);
    res_array = (int*)malloc(sizeof(int)*res_array_size);
    for (int i = 0; i < res_array_size; i +=3)
        res_array[i] = INT_MIN;
    

    // MEM ALLOC IN CUDA
    int cuda_seq1_size = sizeof(char) * (strlen(seq1));
    int cuda_seq2_size = sizeof(char) * seq2_lenght;
    int cuda_weights_arr_size = sizeof(int) * (WEIGHTS);

    cudaStatus = hipMalloc((void**)&cuda_seq1, cuda_seq1_size);
    checkStatus(cudaStatus,"hipMalloc Failed\n");
    cudaStatus = hipMalloc((void**)&cuda_seq2_len, sizeof(int));
    checkStatus(cudaStatus,"hipMalloc Failed\n");
    cudaStatus = hipMalloc((void**)&cuda_seq2, cuda_seq2_size);
    checkStatus(cudaStatus,"hipMalloc Failed\n");
    cudaStatus = hipMalloc((void**)&cuda_weights, cuda_weights_arr_size);
    checkStatus(cudaStatus,"hipMalloc Failed\n");
    cudaStatus = hipMalloc((void**)&cuda_offset_start, sizeof(int));
    checkStatus(cudaStatus,"hipMalloc Failed\n");
    cudaStatus = hipMalloc((void**)&cuda_offset_end, sizeof(int));
    checkStatus(cudaStatus,"hipMalloc Failed\n");
    cudaStatus = hipMalloc((void**)&cuda_res_array, sizeof(int) * res_array_size);
    checkStatus(cudaStatus,"hipMalloc Failed\n");


    // SEND DATA TO GPU
    cudaStatus = hipMemcpy(cuda_seq1, seq1, cuda_seq1_size, hipMemcpyHostToDevice);
    checkStatus(cudaStatus,"hipMemcpy Failed\n");
    cudaStatus = hipMemcpy(cuda_seq2_len, &seq2_lenght, sizeof(int), hipMemcpyHostToDevice);
    checkStatus(cudaStatus,"hipMemcpy Failed\n");
    cudaStatus = hipMemcpy(cuda_seq2, seq2, cuda_seq2_size, hipMemcpyHostToDevice);
    checkStatus(cudaStatus,"hipMemcpy Failed\n");
    cudaStatus = hipMemcpy(cuda_weights, weights, cuda_weights_arr_size, hipMemcpyHostToDevice);
    checkStatus(cudaStatus,"hipMemcpy Failed\n");
    cudaStatus = hipMemcpy(cuda_offset_start, &offset_start, sizeof(int), hipMemcpyHostToDevice);
    checkStatus(cudaStatus,"hipMemcpy Failed\n");
    cudaStatus = hipMemcpy(cuda_offset_end, &offset_end, sizeof(int), hipMemcpyHostToDevice);
    checkStatus(cudaStatus,"hipMemcpy Failed\n");
    cudaStatus = hipMemcpy(cuda_res_array, res_array, sizeof(int) * res_array_size, hipMemcpyHostToDevice);
    checkStatus(cudaStatus,"hipMemcpy Failed\n");
    
    // LUNCH GPU
    thread_score_offset_calc<<<num_of_blocks, THREADS_PER_BLOCK>>>(cuda_seq1, cuda_seq2, cuda_seq2_len, cuda_weights, cuda_offset_start, cuda_offset_end ,cuda_res_array);

    cudaStatus = hipDeviceSynchronize();

    // RECV ALL RESULTS - 1 PER EACH THREAD
    cudaStatus = hipMemcpy(res_array, cuda_res_array, sizeof(int)*res_array_size, hipMemcpyDeviceToHost);
    checkStatus(cudaStatus,"hipMemcpy Failed\n");
    // GET THE BIGGEST SCORE
    optimal_res[SCORE] = INT_MIN; 

    for (int i = 0; i < res_array_size; i += 3)
    {
        if(optimal_res[SCORE] < res_array[i])
        {
            optimal_res[SCORE] = res_array[i];
            optimal_res[OFFSET] = res_array[i+OFFSET];
            optimal_res[MUTANT] = res_array[i+MUTANT];
        }
    }
    
    // FREE 
    hipFree(cuda_seq1);
    hipFree(cuda_seq2);
    hipFree(cuda_weights);
    hipFree(cuda_offset_start);
    hipFree(cuda_offset_end);
    hipFree(cuda_res_array);
    hipFree(cuda_seq2_len);

    return optimal_res;
}
